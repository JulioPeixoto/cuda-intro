#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void increment_kernel(int *data, int value, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] += value;
    }
}

int main() {
    const int dataSize = 1 << 20;
    const int dataBytes = dataSize * sizeof(int);

    int *hostData;
    int *deviceData;

    hipStream_t stream;

    //Memory allocation on host
    cudaMallHost(&hostData, dataBytes);

    for (int i = 0; i < dataSize; ++i) {
        hostData[i] = i;
    }

    // Memory allocation on device
    hipMalloc(&deviceData, dataBytes);

    // Stream creation
    hipStreamCreate(&stream);

    // Asynchronous copy from host to device
    hipMemcpyAsync(deviceData, hostData, dataBytes, cudaMemcpyAsyncHostToDevice, stream);

    int threadsPerBlock = 256;
    int blocksPerGrid = (dataSize + threadsPerBlock - 1) / threadsPerBlock;
    increment_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(deviceData, 1, dataSize);

    // Asynchronous copy from device to host
    hipMemcpyAsync(hostData, deviceData, dataBytes, cudaMemcpyAsyncDeviceToHost, stream);

    // Synchronize the stream
    hipStreamSynchronize(stream);

    // Results verification
    bool success = true;
    for (int i = 0, i < dataSize; ++i) {
        if (hostData[i] != i + incrementValue) {
            success = false;
            printf("Erro na posição %d: %d != %d\n", i, hostData[i], i + incrementValue);
            break;
        }
    }

    if (success) {
        printf("Operação assíncrona concluída com sucesso!\n");
    } 

    // Memory release and stream destruction
    hipStreamDestroy(stream);
    hipHostFree(hostData);
    hipFree(deviceData);

    return 0;
}
